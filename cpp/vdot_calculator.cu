#include "vdot_calculator.h"
#include "mps.h"

#include <iostream>

#define HANDLE_ERROR(x)                                \
{                                                      \
    const auto err = x;	                               \
    if (err != CUTENSORNET_STATUS_SUCCESS) {           \
	std::cout << "Error: " << cutensornetGetErrorString(err) << " in line " << __LINE__ << std::endl; \
	std::exit(1);                                  \
    }                                                  \
};

#define HANDLE_CUDA_ERROR(x)                           \
{                                                      \
    const auto err = x;                                \
    if (err != hipSuccess) {                          \
	std::cout << "Error: " << hipGetErrorString(err) << " in line " << __LINE__ << std::endl; \
	std::exit(1);                                  \
    }                                                  \
};

VdotCalculator::VdotCalculator(hipDataType typeData, cutensornetComputeType_t typeCompute, int numQubits, int physExtent)
{
    typeData_ = typeData;
    typeCompute_ = typeCompute;
    numQubits_ = numQubits;
    physExtent_ = physExtent;

    // create handle
    HANDLE_ERROR(cutensornetCreate(&handle_));

    // create CUDA stream
    HANDLE_CUDA_ERROR(hipStreamCreate(&stream_));
    
    // create workspace and set its memory
    HANDLE_ERROR(cutensornetCreateWorkspaceDescriptor(handle_, &workDesc_));
    // FIXME: for now, hard code this to be similar size to in test program
    // might want to compute it dynamically in future
    workspaceSize_ = 20 * 1024 * 1024;
    HANDLE_CUDA_ERROR(hipMalloc(&workspace_, workspaceSize_));
    HANDLE_ERROR(cutensornetWorkspaceSetMemory(handle_,
					       workDesc_,
					       CUTENSORNET_MEMSPACE_DEVICE,
					       CUTENSORNET_WORKSPACE_SCRATCH,
					       workspace_,
					       workspaceSize_));

    HANDLE_CUDA_ERROR(hipMalloc(&resultGPU_, sizeof(complex_t)));
}

VdotCalculator::~VdotCalculator()
{
    cutensornetDestroyWorkspaceDescriptor(workDesc_);
    cutensornetDestroy(handle_);

    hipFree(workspace_);
    hipFree(resultGPU_);
}

complex_t VdotCalculator::vdot(MatrixProductState& mps1, MatrixProductState& mps2)
{
    // FIXME: promote a lot of the locals here to class members so they don't
    // have to be allocated each time
    // sanity check
    if ((mps1.numQubits_ != numQubits_) ||
	(mps2.numQubits_ != numQubits_) ||
	(mps1.physExtent_ != physExtent_) ||
	(mps2.physExtent_ != physExtent_)) {
	std::cerr << "For vdot, both MPS must have same number of qubits and physical extent!" << std::endl;
	return complex_t(0.0, 0.0);
    }

    std::vector<int32_t> numModesIn;
    std::vector<int64_t*> extentsIn;
    std::vector<int64_t*> stridesIn;
    std::vector<int32_t*> modesIn;

    int64_t extentsOut[1] = { 1 };
    int32_t modesOut[1]; // = { nextMode_; };
    int numModesOut = 1;
    modesOut[0] = 1;
    
    // mps1 tensors
    for (int i = 0; i < mps1.numQubits_; i++) {
	numModesIn.push_back(4);
	
	int64_t* extents = new int64_t[4];
	extents[0] = mps1.extentsPerQubit_[i];
	extents[1] = mps1.extentsPerQubit_[i+1];
	extents[2] = mps1.physExtent_;
	extents[3] = 1;
	extentsIn.push_back(extents);

	int64_t* strides = new int64_t[4];
	strides[3] = 1;
	strides[2] = 1;
	strides[1] = extents[2] * strides[2];
	strides[0] = extents[1] * strides[1];
	stridesIn.push_back(strides);

	int32_t* modes = new int32_t[4];
	modes[0] = mps1.virtualModes_[i];
	modes[1] = mps1.virtualModes_[i+1];
	modes[2] = mps1.physModes_[i];
	modes[3] = modesOut[0];
	modesIn.push_back(modes);
    }

    // mps 2 tensors
    for (int i = 0; i < mps2.numQubits_; i++) {
	numModesIn.push_back(4);

	int64_t* extents = new int64_t[4];
	extents[0] = mps2.extentsPerQubit_[i];
	extents[1] = mps2.extentsPerQubit_[i+1];
	extents[2] = mps2.physExtent_;
	extents[3] = 1;
	extentsIn.push_back(extents);

	int64_t* strides = new int64_t[4];
	strides[3] = 1;
	strides[2] = 1;
	strides[1] = extents[2] * strides[2];
	strides[0] = extents[1] * strides[1];
	stridesIn.push_back(strides);
	
	int32_t* modes = new int32_t[4];
	modes[0] = mps2.virtualModes_[i];
	modes[1] = mps2.virtualModes_[i+1];
	modes[2] = mps1.physModes_[i]; // same physical modes as mps 1
	modes[3] = modesOut[0];
	modesIn.push_back(modes);
    }
    
    cutensornetNetworkDescriptor_t descNet;
    HANDLE_ERROR(cutensornetCreateNetworkDescriptor(handle_, mps1.numQubits_*2,
						    numModesIn.data(),
						    extentsIn.data(),
						    stridesIn.data(),
						    modesIn.data(), nullptr,
						    numModesOut,
						    extentsOut, nullptr, modesOut,
						    typeData_, typeCompute_,
						    &descNet));

    // create optimiser info
    cutensornetContractionOptimizerConfig_t optimizerConfig;
    HANDLE_ERROR( cutensornetCreateContractionOptimizerConfig(handle_, &optimizerConfig) );
    cutensornetContractionOptimizerInfo_t optimizerInfo;
    HANDLE_ERROR( cutensornetCreateContractionOptimizerInfo(handle_, descNet, &optimizerInfo) );
    // leave contraction path implicit, unlike in Pytket. It's slightly faster
    
    HANDLE_ERROR( cutensornetContractionOptimize(handle_, descNet, optimizerConfig,
						 workspaceSize_, optimizerInfo) );
    int64_t numSlices = 0;
    HANDLE_ERROR( cutensornetContractionOptimizerInfoGetAttribute( handle_, optimizerInfo, CUTENSORNET_CONTRACTION_OPTIMIZER_INFO_NUM_SLICES, &numSlices, sizeof(numSlices)) );
								  
    // create contraction plan
    cutensornetContractionPlan_t plan;
    HANDLE_ERROR( cutensornetCreateContractionPlan(handle_, descNet, optimizerInfo,
						   workDesc_, &plan) );
    
    // create slices
    cutensornetSliceGroup_t sliceGroup{};
    HANDLE_ERROR( cutensornetCreateSliceGroupFromIDRange(handle_, 0, numSlices, 1, &sliceGroup) );

    // perform actual contraction
    std::vector<void*> rawDataIn;
    for (int i = 0; i < mps1.numQubits_; i++) {
	rawDataIn.push_back(mps1.qubitTensor_[i]);
    }
    for (int i = 0; i < mps2.numQubits_; i++) {
	rawDataIn.push_back(mps2.qubitTensor_[i]);
    }
    
    HANDLE_ERROR(cutensornetContractSlices(handle_, plan, rawDataIn.data(),
					   resultGPU_, 0, workDesc_,
					   sliceGroup, stream_));

    // copy back result
    complex_t result;
    HANDLE_CUDA_ERROR(hipMemcpy(&result, resultGPU_, sizeof(complex_t),
				 hipMemcpyDeviceToHost));
    
    // free resources
    //delete[] path.data;
    for (int i = 0; i < (mps1.numQubits_ * 2); i++) {
	delete[] extentsIn[i];
	delete[] modesIn[i];
	delete[] stridesIn[i];
    }
    HANDLE_ERROR(cutensornetDestroySliceGroup(sliceGroup));
    HANDLE_ERROR(cutensornetDestroyContractionPlan(plan));
    HANDLE_ERROR(cutensornetDestroyContractionOptimizerInfo(optimizerInfo));
    HANDLE_ERROR(cutensornetDestroyContractionOptimizerConfig(optimizerConfig));
    HANDLE_ERROR(cutensornetDestroyNetworkDescriptor(descNet));

    return result;
}

